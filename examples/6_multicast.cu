#include "hip/hip_runtime.h"
/*
This code uses TMA's 1d tensor load to load
a portion of an array to shared memory and then
change the value in the shared memory and uses TMA's store
to store the portion back to global memory. We print the result
to show the changes are done.
*/

// supress warning about barrier in shared memory on line 32

#include <hip/hip_cooperative_groups.h>
#include <cuda/barrier>
#include <iostream>

#include "matrix_utilities.cuh"
#include "profile_utilities.cuh"
#include "tma.cuh"
#include "tma_tensor_map.cuh"

using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

namespace cg = cooperative_groups;

const int array_size = 128;
const int tile_size = 16;
const int cluster_size = 4; // we use 4 blocks in a cluster

__global__ void __cluster_dims__(cluster_size, 1, 1)
	kernel(const __grid_constant__ CUtensorMap tensor_map, int coordinate,
		   int *result) {
	// cluster metadata
	cg::cluster_group cluster = cg::this_cluster();
	unsigned int clusterBlockRank = cluster.block_rank();

	__shared__ alignas(16) int tile_shared[tile_size];

	// we let the first block in the cluster to load a
	// tile to the shared memory of all 4 blocks
	if (clusterBlockRank == 0) {
		__shared__ barrier bar;

		if (threadIdx.x == 0) {
			init(&bar, blockDim.x);
			cde::fence_proxy_async_shared_cta();
		}
		__syncthreads();

		barrier::arrival_token token;
		if (threadIdx.x == 0) {
			/*
			each bit represents a block in the cluster, starting from the least
			significant bit (the right side)

			here we use block mask 1011, which means
			blocks 0, 1, and 3 will recieve the data from multicast
			whereas block 2 will not

			we will verify this by printing the result
			*/
			uint16_t ctaMask = 0b1011;
			asm volatile(
				"cp.async.bulk.tensor.1d.shared::cluster.global.tile.mbarrier::"
				"complete_tx::bytes.multicast::cluster "
				"[%0], [%1, {%2}], [%3], %4;\n"
				:
				: "r"(static_cast<_CUDA_VSTD::uint32_t>(
					  __cvta_generic_to_shared(tile_shared))),
				  "l"(&tensor_map), "r"(coordinate),
				  "r"(static_cast<_CUDA_VSTD::uint32_t>(
					  __cvta_generic_to_shared(
						  ::cuda::device::barrier_native_handle(bar)))),
				  "h"(ctaMask)
				: "memory");

			token =
				cuda::device::barrier_arrive_tx(bar, 1, sizeof(tile_shared));
		} else {
			token = bar.arrive();
		}

		bar.wait(std::move(token));
	}

	// rest of the clusters needs to wait for cluster 0 to load the data
	cluster.sync();

	// put the results back
	if (clusterBlockRank == 0 && threadIdx.x == 0) {
		for (int i = 0; i < tile_size; ++i) {
			result[clusterBlockRank * tile_size + i] = tile_shared[i];
		}
	}

	if (clusterBlockRank == 1 && threadIdx.x == 0) {
		for (int i = 0; i < tile_size; ++i) {
			result[clusterBlockRank * tile_size + i] = tile_shared[i];
		}
	}

	if (clusterBlockRank == 2 && threadIdx.x == 0) {
		for (int i = 0; i < tile_size; ++i) {
			result[clusterBlockRank * tile_size + i] = tile_shared[i];
		}
	}

	if (clusterBlockRank == 3 && threadIdx.x == 0) {
		for (int i = 0; i < tile_size; ++i) {
			result[clusterBlockRank * tile_size + i] = tile_shared[i];
		}
	}
}

int main() {
	// initialize array and fill it with values
	int h_data[array_size];
	for (size_t i = 0; i < array_size; ++i) {
		h_data[i] = i;
	}

	// print the array before the kernel
	// one tile per line
	print_matrix(h_data, array_size / tile_size, tile_size);

	// transfer array to device
	int *d_data = nullptr;
	hipMalloc(&d_data, array_size * sizeof(int));
	hipMemcpy(d_data, h_data, array_size * sizeof(int),
			   hipMemcpyHostToDevice);

	// create tensor map
	CUtensorMap tensor_map =
		create_1d_tensor_map(array_size, tile_size, d_data);

	// a 2d array that will be used to store the tile loaded to each block
	int *d_result = nullptr;
	hipMalloc(&d_result, tile_size * cluster_size * sizeof(int));

	size_t offset =
		tile_size * 3; // select the second tile of the array to change
	kernel<<<cluster_size, 128>>>(tensor_map, offset, d_result);

	cuda_check_error();

	// transfer the result back to host
	int h_result[tile_size * cluster_size];
	hipMemcpy(h_result, d_result, tile_size * cluster_size * sizeof(int),
			   hipMemcpyDeviceToHost);

	// print the result for each block
	print_matrix(h_result, cluster_size, tile_size);

	hipFree(d_data);

	return 0;
}
