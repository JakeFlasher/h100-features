#include "hip/hip_runtime.h"
/*
This code demonstrates how to use the dense wgmma instructions
to perform matrix multiplication
*/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <cuda/barrier>
#include <hip/hip_fp16.h>
#include <iostream>
#include <mma.h>
#include <random>
#include <stdio.h>

#include "matrix_utilities.cuh"
#include "profile_utilities.cuh"
#include "tma_tensor_map.cuh"
#include "wgmma.cuh"

// Suppress warning about barrier in shared memory
#pragma nv_diag_suppress static_var_with_dynamic_init

using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

const int M = 64;
const int N = 8;
const int K = 16;

const int threads_per_block = 32 * 4; // 4 warps
const int blocks = 1;

__global__ void kernel(const __grid_constant__ CUtensorMap tensor_map, half *B,
					   half *C) {
	// metadata
	const int tid = threadIdx.x;
	const int warp_id = tid / 32;
	const int lane_id = tid % 32;
	const int group_id = lane_id >> 2;
	const int lane_in_group = lane_id & 3;

	__syncthreads();

	__align__(128) __shared__ half A_shared[M * K];
	__align__(16) __shared__ half B_shared[K * N];

	__shared__ barrier bar;

	if (threadIdx.x == 0) {
		init(&bar, blockDim.x);
	}
	__syncthreads();

	// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#asynchronous-multiply-and-accumulate-instruction-wgmma-mma-async
	// 8x8 core blocks, we use one thread here to
	// easy demonstrate the required layout
	if (tid == 0) {
		// load B
		for (int i = 0; i < K; i++) {
			for (int j = 0; j < N; j++) {
				int block_x = i / 8;
				int block_row = i % 8;
				int block_y = j / 8;
				int block_col = j % 8;
				int block_id = block_x * 1 + block_y;
				int offset = block_id * 64 + block_row * 8 + block_col;
				B_shared[offset] = B[i * N + j];
			}
		}
	}

	// Load A
	uint64_t token;
	if (tid == 0) {
		// call the loading api
		cde::cp_async_bulk_tensor_2d_global_to_shared(A_shared, &tensor_map, 0,
													  0, bar);
		token = cuda::device::barrier_arrive_tx(bar, 1, sizeof(A_shared));
	} else {
		token = bar.arrive();
	}

	bar.wait(cuda::std::move(token));

	__syncthreads();

	// create descriptors for the matrices
	GmmaDescriptor desc_a = make_desc_a<half *, 3>(A_shared);
	GmmaDescriptor desc_b = make_desc_b(B_shared);

	// accumulator
	uint32_t c[2] = {};

	// called whenever the accumulator is accessed
	warpgroup_arrive();

	// wgmma.mma_async.sync.aligned.shape.dtype.f16.f16  d, a-desc, b-desc,
	// scale-d, imm-scale-a, imme-scale-b, imm-trans-a, imm-trans-b;
	// wgmma.mma_async.sync.aligned.shape.dtype.f16.f16  d, a, b-desc, scale-d,
	// imm-scale-a, imme-scale-b, imm-trans-b;
	asm volatile("wgmma.mma_async.sync.aligned.m64n8k16.f16.f16.f16 "
				 "{%0, %1}, " // accumulator
				 "%2, %3, "	  // matrix a descriptor
				 "1, "		  // 0 => D = A*B, 1 => D = D + A*B
				 "1, 1, " // 0 => no scaling, 1 => scaling, scaling means times
						  // -1 to a or b
				 "0, 1;" // transpose a and b, 0 => no transpose, 1 => transpose
				 : "+r"(c[0]), "+r"(c[1])
				 : "l"(desc_a), "l"(desc_b));

	// commit, start the computation
	warpgroup_commit_batch();

	// wait for the previous commit to finish
	warpgroup_wait<0>();

	// thread fence needed for async operations
	__threadfence();

	warpgroup_arrive();

	uint32_t *C_ptr = reinterpret_cast<uint32_t *>(C);

	int offset1 = warp_id * 16 * 4 + group_id * 4 + lane_in_group;
	int offset2 = warp_id * 16 * 4 + (group_id + 8) * 4 + lane_in_group;

	// write back to global memory
	C_ptr[offset1] = c[0];
	C_ptr[offset2] = c[1];
}

int main() {

	half *d_C;
	half h_C[M * N];
	half h_CPU[M * N];
	half h_A[M * K];
	half h_B[K * N];

	fill_fixed(h_C, M, N, 0);

	fill_random(h_A, M, K);
	fill_random(h_B, K, N);

	half *d_A, *d_B;

	hipMalloc((void **)&d_A, M * K * sizeof(half));
	hipMalloc((void **)&d_B, K * N * sizeof(half));
	hipMalloc((void **)&d_C, M * N * sizeof(half));

	hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);

	CUtensorMap tensor_map = create_2d_tensor_map_half<1>(M, K, M, K, d_A);

	kernel<<<blocks, threads_per_block>>>(tensor_map, d_B, d_C);

	cuda_check_error();

	hipMemcpy(h_C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);

	// print_matrix(h_C, M, N);

	CPU_gemm(h_A, h_B, h_CPU, M, N, K);

	compare_matrices(h_CPU, h_C, M, N);

	// print_differnce(h_C, h_CPU, M, N, 0.0f);

	return 0;
}
