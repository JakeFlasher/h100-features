#include "hip/hip_runtime.h"
/*
This code demonstrates how to use the dense wgmma instructions
to perform matrix multiplication
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <assert.h>
#include <hip/hip_fp16.h>
#include <random>
#include <iostream>

#include "matrix_utilities.cuh"
#include "profile_utilities.cuh"
#include "wgmma.cuh"

const int M = 64;
const int N = 8;
const int K = 16;

const int threads_per_block = 32 * 4; // 4 warps
const int blocks = 1;

__global__ void kernel(half *A, half *B, half *C)
{
  // metadata
  const int tid = threadIdx.x;
  const int warp_id = tid / 32;
  const int lane_id = tid % 32;
  const int group_id = lane_id >> 2;
  const int lane_in_group = lane_id & 3;

  __syncthreads();

  __align__(16) __shared__ half A_shared[M * K];
  __align__(16) __shared__ half B_shared[K * N];

  __align__(16) __shared__ half buffer[2 * 64];

  // https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#asynchronous-multiply-and-accumulate-instruction-wgmma-mma-async
  // 8x8 core blocks, we use one thread here to
  // easy demonstrate the required layout
  if (tid == 0)
  {
    for (int i = 0; i < M; i++)
    {
      for (int j = 0; j < K; j++)
      {
        int block_x = i / 8;
        int block_row = i % 8;
        int block_y = j / 8;
        int block_col = j % 8;
        int block_id = block_x * 2 + block_y;
        int offset = block_id * 64 + block_row * 8 + block_col;
        A_shared[offset] = A[i * K + j];
      }
    }
    
    // swizzle A
    for (int pair = 0; pair < 8; pair++) {
        
        for (int i = 0; i < 8; i++) {
            if (i % 2 == 0) {
                for (int j = 0; j < 8; j++) {
                    buffer[i * 8 + j] = A_shared[pair * 128 + i / 2 * 8 + j];
                }
            }
            else {
                for (int j = 0; j < 8; j++) {
                    buffer[i * 8 + j] = A_shared[pair * 128 + 64 + i / 2 * 8 + j];
                }
            }
        }
        
        for (int i = 0; i < 8; i++) {
            if (i % 2 == 0) {
                for (int j = 0; j < 8; j++) {
                    buffer[64 + i * 8 + j] = A_shared[pair * 128 + 64 + i / 2 * 8 + j];
                }
            }
            else {
                for (int j = 0; j < 8; j++) {
                    buffer[64 + i * 8 + j] = A_shared[pair * 128 + i / 2 * 8 + j];
                }
            }
        }
        
        // write back to A_shared
        for (int row = 0; row < 16; row ++) {
            for (int col = 0; col < 8; col++) {
                A_shared[pair * 128 + row * 8 + col] = buffer[row * 8 + col];
            }
        }
    }

    for (int i = 0; i < K; i++)
    {
      for (int j = 0; j < N; j++)
      {
        int block_x = i / 8;
        int block_row = i % 8;
        int block_y = j / 8;
        int block_col = j % 8;
        int block_id = block_x * 1 + block_y;
        int offset = block_id * 64 + block_row * 8 + block_col;
        B_shared[offset] = B[i * N + j];
      }
    }
  }

  __syncthreads();

  // create descriptors for the matrices
  GmmaDescriptor desc_a = make_desc_a(A_shared);
  GmmaDescriptor desc_b = make_desc_b(B_shared);

  // accumulator
  uint32_t c[2] = {};

  // called whenever the accumulator is accessed
  warpgroup_arrive();

  // wgmma.mma_async.sync.aligned.shape.dtype.f16.f16  d, a-desc, b-desc, scale-d, imm-scale-a, imme-scale-b, imm-trans-a, imm-trans-b;
  // wgmma.mma_async.sync.aligned.shape.dtype.f16.f16  d, a, b-desc, scale-d, imm-scale-a, imme-scale-b, imm-trans-b;
  asm volatile("wgmma.mma_async.sync.aligned.m64n8k16.f16.f16.f16 "
               "{%0, %1}, " // accumulator
               "%2, %3, "   // matrix a descriptor
               "1, "        // 0 => D = A*B, 1 => D = D + A*B
               "1, 1, "     // 0 => no scaling, 1 => scaling, scaling means times -1 to a or b
               "0, 1;"      // transpose a and b, 0 => no transpose, 1 => transpose
               : "+r"(c[0]), "+r"(c[1])
               : "l"(desc_a), "l"(desc_b));

  // commit, start the computation
  warpgroup_commit_batch();

  // wait for the previous commit to finish
  warpgroup_wait<0>();

  // thread fence needed for async operations
  __threadfence();

  warpgroup_arrive();

  uint32_t *C_ptr = reinterpret_cast<uint32_t *>(C);

  int offset1 = warp_id * 16 * 4 + group_id * 4 + lane_in_group;
  int offset2 = warp_id * 16 * 4 + (group_id + 8) * 4 + lane_in_group;

  // write back to global memory
  C_ptr[offset1] = c[0];
  C_ptr[offset2] = c[1];
}

int main()
{

  half *d_C;
  half h_C[M * N];
  half h_CPU[M * N];
  half h_A[M * K];
  half h_B[K * N];

  fill_fixed(h_C, M, N, 0);

  fill_random(h_A, M, K);
  fill_random(h_B, K, N);

  half *d_A, *d_B;

  hipMalloc((void **)&d_A, M * K * sizeof(half));
  hipMalloc((void **)&d_B, K * N * sizeof(half));
  hipMalloc((void **)&d_C, M * N * sizeof(half));

  hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);

  kernel<<<blocks, threads_per_block>>>(d_A, d_B, d_C);

  cuda_check_error();

  hipMemcpy(h_C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);

  // print_matrix(h_C, M, N);

  CPU_gemm(h_A, h_B, h_CPU, M, N, K);

  compare_matrices(h_CPU, h_C, M, N);

  // print_differnce(h_C, h_CPU, M, N, 0.0f);

  return 0;
}
