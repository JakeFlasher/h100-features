#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>

__global__ void __cluster_dims__(2, 1, 1) cluster_kernel()
{
  // printf("blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x, threadIdx.x);

  __shared__ int smem[32];
  namespace cg = cooperative_groups;
  int tid = cg::this_grid().thread_rank();

  cg::cluster_group cluster = cg::this_cluster();
  unsigned int clusterBlockRank = cluster.block_rank();
  int cluster_size = cluster.dim_blocks().x;

  smem[tid] = blockIdx.x + threadIdx.x;

  cluster.sync();

  int *other_block_smem = cluster.map_shared_rank(smem, 1 - clusterBlockRank);

  int value = other_block_smem[tid];

  // print the value
  printf("blockIdx.x: %d, threadIdx.x: %d, value: %d\n", blockIdx.x, threadIdx.x, value);
}

int main()
{

  // two blocks in a cluster
  cluster_kernel<<<2, 32>>>();

  hipDeviceSynchronize();

  // check for kernel errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("CUDA error: %s\n", hipGetErrorString(err));
  }
}
