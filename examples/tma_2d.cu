#include "hip/hip_runtime.h"
// This code uses TMA's 2d load to load a matrix's tile to
// shared memory and then change the value in the
// shared memory and uses TMA's store to store the
// tile back to global memory. We print the result matrix to prove the
// changes are done

#include <cuda/barrier>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "test_macros.cuh"
#include "tma_tensor_map.cuh"
#include "matrix_utilities.cuh"
#include "tma.cuh"

// Suppress warning about barrier in shared memory
TEST_NV_DIAG_SUPPRESS(static_var_with_dynamic_init)

using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

constexpr size_t M = 64; // Number of rows of matrix
constexpr size_t K = 32; // Number of columns of matrix
constexpr size_t gmem_len = M * K;

constexpr int m = 16; // subtile rows
constexpr int k = 8;  // subtile columns

static constexpr int buf_len = k * m;

__global__ void test(const __grid_constant__ CUtensorMap global_fake_tensor_map, int base_i, int base_j)
{
  __shared__ alignas(128) int smem_buffer[buf_len];
  __shared__ barrier bar;

  if (threadIdx.x == 0)
  {
    init(&bar, blockDim.x);
  }
  __syncthreads();

  // Load data:
  uint64_t token;
  if (threadIdx.x == 0)
  {
    // just to demonstrate prefetch
    // copy_async_2d_prefetch(global_fake_tensor_map, base_j, base_i);
    // call the loading api
    cde::cp_async_bulk_tensor_2d_global_to_shared(smem_buffer, &global_fake_tensor_map, base_j, base_i, bar);
    token = cuda::device::barrier_arrive_tx(bar, 1, sizeof(smem_buffer));
  }
  else
  {
    token = bar.arrive();
  }

  bar.wait(cuda::std::move(token));

  __syncthreads();

  // Update subtile, + 1
  for (int i = threadIdx.x; i < buf_len; i += blockDim.x)
  {
    smem_buffer[i] += 1;
  }

  cde::fence_proxy_async_shared_cta();
  __syncthreads();

  // Write back to global memory:
  if (threadIdx.x == 0)
  {
    cde::cp_async_bulk_tensor_2d_shared_to_global(&global_fake_tensor_map, base_j, base_i, smem_buffer);
    cde::cp_async_bulk_commit_group();
    cde::cp_async_bulk_wait_group_read<0>();
  }
  __threadfence();
  __syncthreads();
}

int main()
{
  // fill the host matrix
  int host_tensor[gmem_len];
  fill_tilewise(host_tensor, M, K, m, k);

  print_matrix(host_tensor, M, K);

  // copy host matrix to device
  int *tensor_ptr = nullptr;
  hipMalloc(&tensor_ptr, gmem_len * sizeof(int));
  hipMemcpy(tensor_ptr, host_tensor, gmem_len * sizeof(int), hipMemcpyHostToDevice);

  // create tensor map for the matrix
  CUtensorMap tensor_map = create_2d_tensor_map(M, K, m, k, tensor_ptr);

  // launch kernel, select a tile coordinate
  int coordinate_m = 0;
  int coordinate_k = 16;
  test<<<1, 128>>>(tensor_map, coordinate_m, coordinate_k);

  hipDeviceSynchronize();

  // check for kernel errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("CUDA error: %s\n", hipGetErrorString(err));
  }

  // copy device matrix to host
  int host_gmem_tensor[gmem_len];
  hipMemcpy(host_gmem_tensor, tensor_ptr, gmem_len * sizeof(int), hipMemcpyDeviceToHost);

  // verify the results
  print_matrix(host_gmem_tensor, M, K);

  return 0;
}