#include "hip/hip_runtime.h"
/*
This code demonstrates how to use the sparse wgmma instructions
to perform matrix multiplication

Sparse means matrix A follows a 2:4 format
*/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <mma.h>
#include <random>
#include <stdio.h>

#include "matrix_utilities.cuh"
#include "profile_utilities.cuh"
#include "wgmma.cuh"

const int M = 64;
const int N = 8;
const int K = 32;

// 2:4 format
const int K_A = 16;

const int threads_per_block = 32 * 4; // 4 warps
const int blocks = 1;

__global__ void kernel(half *A, half *B, half *C, u_int32_t *metadata_array) {
	const int tid = threadIdx.x;
	const int warp_id = tid / 32;
	const int lane_id = tid % 32;
	const int group_id = lane_id >> 2;
	const int lane_in_group = lane_id & 3;
	const int lane_in_work_group = lane_in_group % 2;

	__align__(16) __shared__ half A_shared[M * K_A];
	__align__(16) __shared__ half B_shared[K * N];

	// use one thread to load so it's easier to tell the layout
	// refer to the ptx menu for the layout of the shared memory
	if (tid == 0) {
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < K_A; j++) {
				int block_x = i / 8;
				int block_row = i % 8;
				int block_y = j / 8;
				int block_col = j % 8;
				int block_id = block_x * 2 + block_y;
				int offset = block_id * 64 + block_row * 8 + block_col;
				A_shared[offset] = A[i * K_A + j];
			}
		}

		for (int i = 0; i < K; i++) {
			for (int j = 0; j < N; j++) {
				int block_x = i / 8;
				int block_row = i % 8;
				int block_y = j / 8;
				int block_col = j % 8;
				int block_id = block_x * 1 + block_y;
				int offset = block_id * 64 + block_row * 8 + block_col;
				B_shared[offset] = B[i * N + j];
			}
		}
	}

	__syncthreads();

	// load metadata
	u_int32_t metadata;
	uint metadata_offset = warp_id * 16 + lane_in_work_group * 8 + group_id;
	metadata = metadata_array[metadata_offset];

	__syncthreads();

	// create descriptors
	GmmaDescriptor desc_a = make_desc<half *, 8, 16, 0>(A_shared);
	GmmaDescriptor desc_b = make_desc<half *, 8, 16, 0>(B_shared);

	// accumulator
	uint32_t c[2] = {};

	warpgroup_arrive();

	asm volatile("wgmma.mma_async.sp.sync.aligned.m64n8k32.f16.f16.f16 "
				 "{%0, %1}, " // c
				 "%2, %3, "	  // desc A, B
				 "%4, "		  // meta
				 "0, "		  // thread selection
				 "1, "		  // scale D
				 "%7, %8, "	  // +/- scale A, B
				 "%9, %10;"	  // transpose A, B
				 : "+r"(c[0]), "+r"(c[1])
				 : "l"(desc_a), "l"(desc_b),
				   "r"(metadata),	// metadata
				   "r"(0),			// thread selection
				   "r"(1),			// scale D
				   "n"(1), "n"(1),	// +- scale A, B
				   "n"(0), "n"(1)); // transpose A, B

	// commit, start the computation
	warpgroup_commit_batch();

	// wait for the previous commit to finish
	warpgroup_wait<0>();

	// thread fence needed for async operations
	__threadfence();

	warpgroup_arrive();

	// store the result
	uint32_t *C_ptr = reinterpret_cast<uint32_t *>(C);

	int offset1 = warp_id * 16 * 4 + group_id * 4 + lane_in_group;
	int offset2 = warp_id * 16 * 4 + (group_id + 8) * 4 + lane_in_group;

	C_ptr[offset1] = c[0];
	C_ptr[offset2] = c[1];
}

int main() {

	half *d_C;
	half h_C[M * N];
	half h_CPU[M * N];
	half h_A[M * K];
	half h_A2[M * K_A];
	half h_B[K * N];

	fill_24(h_A, M, K);
	fill_random(h_B, K, N);

	// print_matrix(h_A, M, K);

	// extract the non-zeros in each 2:4 tile to a compressed matrix A2
	compress24(h_A, h_A2, M, K);

	// print_matrix(h_A2, M, K_A);

	half *d_A, *d_B;

	hipMalloc((void **)&d_A, M * K_A * sizeof(half));
	hipMalloc((void **)&d_B, K * N * sizeof(half));
	hipMalloc((void **)&d_C, M * N * sizeof(half));

	hipMemcpy(d_A, h_A2, M * K_A * sizeof(half), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);

	int metadata_size = (M / 16) * (K / 16) * 8;

	u_int32_t *metadata_array = new u_int32_t[metadata_size];
	inspect_metadata(h_A, metadata_array, M, K);

	u_int32_t *d_metadata;
	hipMalloc((void **)&d_metadata, metadata_size * sizeof(u_int32_t));
	hipMemcpy(d_metadata, metadata_array, metadata_size * sizeof(u_int32_t),
			   hipMemcpyHostToDevice);

	kernel<<<blocks, threads_per_block>>>(d_A, d_B, d_C, d_metadata);

	cuda_check_error();

	hipMemcpy(h_C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);

	print_matrix<5>(h_A2, M, K_A);

	CPU_gemm(h_A, h_B, h_CPU, M, N, K);

	compare_matrices(h_CPU, h_C, M, N);

	// print_differnce(h_CPU, h_C, M, N, 0);

	return 0;
}
