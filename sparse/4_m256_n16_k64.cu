#include "hip/hip_runtime.h"
/*
This code demonstrates how to use the sparse wgmma instructions
to perform matrix multiplication

Sparse means matrix A follows a 2:4 format
*/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <mma.h>
#include <random>
#include <stdio.h>
#include <cuda/barrier>

#include "matrix_utilities.cuh"
#include "profile_utilities.cuh"
#include "tma_tensor_map.cuh"
#include "wgmma.cuh"

#pragma nv_diag_suppress static_var_with_dynamic_init

const int M = 256;
const int N = 16;
const int K = 64;

// 2:4 format
const int K_A = 32;

const int threads_per_block = 32 * 4; // 4 warps
const int blocks = 1;

using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

__device__ void MMA_SP_WRAPPER(uint32_t * c, GmmaDescriptor desc_a, GmmaDescriptor desc_b, uint32_t metadata) {
	asm volatile("wgmma.mma_async.sp.sync.aligned.m64n8k32.f16.f16.f16 "
				 "{%0, %1, %2, %3}, " // c
				 "%4, %5, "	  // desc A, B
				 "%6, "		  // meta
				 "0, "		  // thread selection
				 "1, "		  // scale D
				 "%9, %10, "	  // +/- scale A, B
				 "%11, %12;"	  // transpose A, B
				 : "+r"(c[0]), "+r"(c[1]), "+r"(c[2]), "+r"(c[3])
				 : "l"(desc_a), "l"(desc_b),
				   "r"(metadata),	// metadata
				   "r"(0),			// thread selection
				   "r"(1),			// scale D
				   "n"(1), "n"(1),	// +- scale A, B
				   "n"(0), "n"(1)); // transpose A, B
}


__global__ void kernel(
                        const __grid_constant__ CUtensorMap tensor_map_a,
                        const __grid_constant__ CUtensorMap tensor_map_b,
                        half *C,
                        u_int32_t *metadata_array) {

	const int tid = threadIdx.x;
	const int warp_id = tid / 32;
	const int lane_id = tid % 32;
	const int group_id = lane_id >> 2;
	const int lane_in_group = lane_id & 3;
	const int lane_in_work_group = lane_in_group % 2;

	__align__(128) __shared__ half A_shared[M * K_A];
	__align__(16) __shared__ half B_shared[K * N];

	__shared__ barrier bar;

	if (threadIdx.x == 0) {
		init(&bar, blockDim.x);
	}
	__syncthreads();

	uint64_t token;
	if (tid == 0) {
		// call the loading api
		cde::cp_async_bulk_tensor_2d_global_to_shared(A_shared, &tensor_map_a, 0,
													  0, bar);
		cde::cp_async_bulk_tensor_2d_global_to_shared(B_shared, &tensor_map_b,
													  0, 0, bar);
		token = cuda::device::barrier_arrive_tx(bar, 1, sizeof(A_shared) + sizeof(B_shared));
	} else {
		token = bar.arrive();
	}

	bar.wait(cuda::std::move(token));
	__syncthreads();


	u_int32_t metadata;
	uint metadata_offset;
	GmmaDescriptor desc_a, desc_b;

	// divide the 256x64 of A into 4 64x32 tiles and multiply them with the B divided into 2 32x16 tiles
	// accumulator
	uint32_t c[4][4] = {};
	
	desc_b = make_desc<half *, 8, 16, 0>(B_shared);
	#pragma unroll
	for (int m2 = 0; m2 < 4; m2++) {
		warpgroup_arrive();
		desc_a = make_desc<half *, 8, 32, 2>(A_shared + m2 * 64 * K_A);
		metadata_offset = m2 * 8 * 4 * 4 + warp_id * 8 * 4 + lane_in_work_group * 8 + group_id;
		metadata = metadata_array[metadata_offset];
		MMA_SP_WRAPPER(c[m2], desc_a, desc_b, metadata);
	}
	
	desc_b = make_desc<half *, 8, 16, 0>(B_shared + 32 * N);
	#pragma unroll
	for (int m2 = 0; m2 < 4; m2++) {
		warpgroup_arrive();
		desc_a = make_desc<half *, 8, 32, 2>(A_shared + m2 * 64 * K_A + K_A / 2);
		metadata_offset = m2 * 8 * 4 * 4 + warp_id * 8 * 4 + 8 * 2 + lane_in_work_group * 8 + group_id;
		metadata = metadata_array[metadata_offset];
		MMA_SP_WRAPPER(c[m2], desc_a, desc_b, metadata);
	}

	// commit, start the computation
	warpgroup_commit_batch();

	// wait for the previous commit to finish
	warpgroup_wait<0>();

	// thread fence needed for async operations
	__threadfence();

	warpgroup_arrive();

	// store the result
	uint32_t *C_ptr = reinterpret_cast<uint32_t *>(C);
	
	for (int m2 = 0; m2 < 4; m2++) {
		int offset1 = m2 * 64 * N / 2 + warp_id * 16 * N / 2 + group_id * N / 2 + lane_in_group;
		int offset2 = m2 * 64 * N / 2 + warp_id * 16 * N / 2 + (group_id + 8) * N / 2 + lane_in_group;
		C_ptr[offset1] = c[m2][0];
		C_ptr[offset2] = c[m2][1];
	}
}

int main() {

	half *d_C;
	half h_C[M * N];
	half h_CPU[M * N];
	half h_A[M * K];
	half h_A2[M * K_A];
	half h_B[K * N];

	fill_24(h_A, M, K);
	fill_random(h_B, K, N);

	// extract the non-zeros in each 2:4 tile to a compressed matrix A2
	compress24(h_A, h_A2, M, K);

	// print_matrix(h_A2, M, K_A);

	half *d_A, *d_B;

	hipMalloc((void **)&d_A, M * K_A * sizeof(half));
	hipMalloc((void **)&d_B, K * N * sizeof(half));
	hipMalloc((void **)&d_C, M * N * sizeof(half));

	hipMemcpy(d_A, h_A2, M * K_A * sizeof(half), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);

	int metadata_size = (M / 16) * (K / 16) * 8;

	u_int32_t *metadata_array = new u_int32_t[metadata_size];
	inspect_metadata(h_A, metadata_array, M, K);

	u_int32_t *d_metadata;
	hipMalloc((void **)&d_metadata, metadata_size * sizeof(u_int32_t));
	hipMemcpy(d_metadata, metadata_array, metadata_size * sizeof(u_int32_t),
			   hipMemcpyHostToDevice);

	CUtensorMap tensor_map_a = create_2d_tensor_map<half, CU_TENSOR_MAP_DATA_TYPE_FLOAT16, CU_TENSOR_MAP_SWIZZLE_64B>(M, K_A, M, K_A, d_A);
	CUtensorMap tensor_map_b = create_2d_tensor_map<half, CU_TENSOR_MAP_DATA_TYPE_FLOAT16, CU_TENSOR_MAP_SWIZZLE_32B>(K, N, K, N, d_B);

	kernel<<<blocks, threads_per_block>>>(tensor_map_a, tensor_map_b, d_C, d_metadata);

	cuda_check_error();

	hipMemcpy(h_C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);

	// print_matrix<5>(h_A2, M, K_A);

	CPU_gemm(h_A, h_B, h_CPU, M, N, K);

	compare_matrices(h_CPU, h_C, M, N);

	// print_differnce(h_CPU, h_C, M, N, 0);

	return 0;
}
